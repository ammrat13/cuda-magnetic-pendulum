#include "hip/hip_runtime.h"
#include "kern_gpu.cuh"


__device__ const unsigned int ITERS = 200000;
__device__ const float STEP = .0002;


__device__ inline float4 operator*(const float& a, const float4& b) {
    return make_float4(
        a*b.x,
        a*b.y,
        a*b.z,
        a*b.w
    );
}

__device__ inline float4 operator+(const float4& a, const float4& b) {
    return make_float4(
        a.x + b.x,
        a.y + b.y,
        a.z + b.z,
        a.w + b.w
    );
}


__global__ void kern::gpu::compute_gpu(
    kern::RawState st,
    size_t resolution,
    kern::Vec2D top,
    kern::Vec2D bot
) {

    unsigned int rowIdx = blockIdx.y;
    unsigned int colIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if(colIdx >= resolution) {
        return;
    }

    float rowA = (float) rowIdx / (float) resolution;
    float colA = (float) colIdx / (float) resolution;

    float4 state = make_float4(
        (1-colA)*top.x + colA*bot.x,
        (1-rowA)*top.y + rowA*bot.y,
        0.0,
        0.0
    );

    for(unsigned int i = 0; i < ITERS; i++) {
        float4 k1 = kern::gpu::state_dt(state);
        float4 k2 = kern::gpu::state_dt(state + STEP/2 * k1);
        float4 k3 = kern::gpu::state_dt(state + STEP/2 * k2);
        float4 k4 = kern::gpu::state_dt(state + STEP * k3);
        state = state + STEP/6 * (k1 + 2*k2 + 2*k3 + k4);
    }

    st.data[rowIdx * st.pitch/sizeof(float4) + colIdx] = state;
}

__device__ float4 kern::gpu::state_dt(float4 state) {
    float4 top_inv_sq = kern::gpu::dts::inv_sq(1.5, 0.15, make_float2(0.0, 0.5), state);
    float4 bot_inv_sq = kern::gpu::dts::inv_sq(1.5, 0.15, make_float2(0.0,-0.5), state);
    float4 mid_spring = kern::gpu::dts::spring(0.5, make_float2(0.0, 0.0), state);

    float4 frict_force = kern::gpu::dts::frict(0.1, state);

    return make_float4(state.z, state.w, 0.0, 0.0)
        + top_inv_sq
        + bot_inv_sq
        + mid_spring
        + frict_force;
}


__device__ float4 kern::gpu::dts::inv_sq(float g, float off, float2 center, float4 state) {
    float2 d = make_float2(state.x-center.x, state.y-center.y);
    float mag = pow(d.x*d.x + d.y*d.y + off*off, -1.5);
    return make_float4(0.0, 0.0, -g*mag*d.x, -g*mag*d.y);
}

__device__ float4 kern::gpu::dts::spring(float k, float2 center, float4 state) {
    float2 d = make_float2(state.x-center.x, state.y-center.y);
    return make_float4(0.0, 0.0, -k*d.x, -k*d.y);
}

__device__ float4 kern::gpu::dts::frict(float m, float4 state) {
    return make_float4(0.0, 0.0, -m*state.z, -m*state.w);
}
