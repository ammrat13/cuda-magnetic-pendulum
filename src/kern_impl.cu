#include "hip/hip_runtime.h"
#include "kern_impl.cuh"
#include "kern_gpu.cuh"


kern::Kern::KernImpl::KernImpl(kern::Params p): p{p}, first_call{true} {

    this->hos_state.pitch = this->p.resolution * sizeof(float4);
    this->hos_state.data = new float4[this->p.pixels()];

    auto alloc_res = hipMallocPitch(
        &this->dev_state.data,
        &this->dev_state.pitch,
        this->p.resolution * sizeof(float4),
        this->p.resolution
    );
    if(alloc_res != hipSuccess) {
        throw;
    }
}

kern::Kern::KernImpl::~KernImpl() {
    delete[] this->hos_state.data;
    hipFree(this->dev_state.data);
}


std::unique_ptr<const kern::State> kern::Kern::KernImpl::getState() const {

    auto ret = std::make_unique<kern::State>(this->p.pixels());

    for(size_t r = 0; r < this->p.resolution; r++) {
        for(size_t c = 0; c < this->p.resolution; c++) {
            float4 cur = this->hos_state.data[
                r * this->hos_state.pitch/sizeof(float4) + c
            ];
            ret[r * this->p.resolution + c] = {
                {cur.x, cur.y},
                {cur.z, cur.w}
            };
        }
    }

    return ret;
}


void kern::Kern::KernImpl::compute(size_t iters) {
    const size_t parallelism = 256;

    const dim3 num_blocks(
        (this->p.resolution + parallelism - 1) / parallelism,
        this->p.resolution,
        1
    );
    const dim3 th_per_blk(parallelism, 1, 1);

    kern::gpu::compute_gpu<<<num_blocks, th_per_blk>>>(
        this->dev_state,
        this->p,
        iters,
        this->first_call
    );

    auto memcpy_res = hipMemcpy2D(
        this->hos_state.data,
        this->hos_state.pitch,
        this->dev_state.data,
        this->dev_state.pitch,
        this->p.resolution * sizeof(float4),
        this->p.resolution,
        hipMemcpyDeviceToHost
    );
    if(memcpy_res != hipSuccess) {
        throw;
    }

    this->first_call = false;
}
