#include "hip/hip_runtime.h"
#include "kern_impl.cuh"
#include "kern_gpu.cuh"


kern::Kern::KernImpl::KernImpl(kern::Params p): p{p}, first_call{true} {

    this->hos_state.pitch = this->p.resolution * sizeof(float4);
    this->hos_state.data = new float4[this->p.pixels()];

    hipError_t alloc_res = hipMallocPitch(
        &this->dev_state.data,
        &this->dev_state.pitch,
        this->p.resolution * sizeof(float4),
        this->p.resolution
    );
    if(alloc_res != hipSuccess) {
        throw;
    }
}

kern::Kern::KernImpl::~KernImpl() {
    delete[] this->hos_state.data;
    hipFree(this->dev_state.data);
}


std::unique_ptr<const kern::StateElem[]> kern::Kern::KernImpl::getState() const {

    std::unique_ptr<kern::StateElem[]> ret(
        new kern::StateElem[this->p.pixels()]
    );

    for(size_t r = 0; r < this->p.resolution; r++) {
        for(size_t c = 0; c < this->p.resolution; c++) {
            float4 cur = this->hos_state.data[
                r * this->hos_state.pitch/sizeof(float4) + c
            ];
            ret[r * this->p.resolution + c] = {
                {cur.x, cur.y},
                {cur.z, cur.w}
            };
        }
    }

    return ret;
}


void kern::Kern::KernImpl::compute(size_t iters) {
    const size_t parallelism = 256;

    const dim3 num_blocks(
        (this->p.resolution + parallelism - 1) / parallelism,
        this->p.resolution,
        1
    );
    const dim3 th_per_blk(parallelism, 1, 1);

    kern::gpu::compute_gpu<<<num_blocks, th_per_blk>>>(
        this->dev_state,
        this->p,
        iters,
        this->first_call
    );

    hipError_t memcpy_res = hipMemcpy2D(
        this->hos_state.data,
        this->hos_state.pitch,
        this->dev_state.data,
        this->dev_state.pitch,
        this->p.resolution * sizeof(float4),
        this->p.resolution,
        hipMemcpyDeviceToHost
    );
    if(memcpy_res != hipSuccess) {
        throw;
    }

    this->first_call = false;
}
